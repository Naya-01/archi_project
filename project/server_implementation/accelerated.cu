extern "C" {

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdint.h>
#include <stdio.h>
#define CHECK_CUDA_INIT(err) if ((err)) {fprintf(stderr, "Could not initialize CUDA ! cudaError=%d\n", (err)); return err;}
#ifndef CUDA_BLOCK_SIZE
#define CUDA_BLOCK_SIZE 16
#endif
    __global__ void cudaKernel_matrixMul(int *A, int *B, int *C, int N) {
        //TODO perform matrix multiplication on the CUDA executor
        //This is a good validation step to ensure you are on the right track for the project
    }
    __global__ void cudaKernel_encrypt() {
        //TODO perform the GPU computation for the full encryption algorithm
        //You are free to change the parameters as you see fit. Or even split the work in multiple kernels.
    }
    void cu_matrix_mul(int *matrix1, int *matrix2, int *result, uint32_t K) {
        int *dev_A, *dev_B, *dev_res;
        hipMalloc((void **)&dev_A, K * K * sizeof(int));
        hipMalloc((void **)&dev_B, K * K * sizeof(int));
        hipMalloc((void **)&dev_res, K * K * sizeof(int));

        // Copy matrices A and B from host to device
        hipMemcpy(dev_A, matrix1, K * K * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_B, matrix2, K * K * sizeof(int), hipMemcpyHostToDevice);

        // Define grid and block dimensions
        dim3 blockSize(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE); // Define the block size (e.g., 16x16 threads per block)
        dim3 gridSize((K + blockSize.x - 1) / blockSize.x, (K + blockSize.y - 1) / blockSize.y);

        // Launch the kernel
        cudaKernel_matrixMul<<<gridSize, blockSize>>>(dev_A, dev_B, dev_res, K);

        // Copy the result matrix C from device to host
        hipMemcpy(result, dev_res, K * K * sizeof(int), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(dev_A);
        hipFree(dev_B);
        hipFree(dev_res);
    }

    void cu_encrypt(int n_rounds, int key_size, int * key, uint32_t K, int * matrix, int * mult_mat) {
        //TODO perform CUDA allocations, kernel invocation, etc for the 
        //full encryption algorithm    
    }

    int cu_init() {
        hipError_t err = cudaInitDevice(0, 0, 0);
        CHECK_CUDA_INIT(err)
        err = hipSetDevice(0);
        CHECK_CUDA_INIT(err)
        err = hipFree(0);
        CHECK_CUDA_INIT(err)
        return err;
    }
    
    int cu_exit() {
        return 0;
    }

}
